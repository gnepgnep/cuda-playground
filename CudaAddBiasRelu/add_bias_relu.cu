#include "hip/hip_runtime.h"
#include "use_tensor.h"

template <typename T>
__global__ void AddBiasRelu0Kernel(const T* x, const T* bias, T* y, const int N) {
    auto bid = blockIdx.x;
    auto tid = threadIdx.x;
    for (auto id = tid; id < N; id += blockDim.x) {
        float v = (float)x[bid * N + id] + (float)bias[id];
        v = fmaxf(v, 0);
        y[bid * N + id] = (T)v;
    }
}

int AddBiasRelu0(const float* x, const float* bias, float* y, 
                        const int B, const int N, hipStream_t stream) {
    dim3 grid(B);
    dim3 block(std::min(N, 1024));
    AddBiasRelu0Kernel<float><<<grid, block, 0, stream>>>(x, bias, y, N);
    if (hipGetLastError() != hipSuccess){
        printf("lauch kernel failed");
        return -1;
    }
    return 0;
}

template <typename T>
__global__ void AddBiasReluKernel1(const T* x, const T* bias, T* y, const int N, const int Num) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < Num) {
        int bias_id = id % N;
        float v = (float)x[id] + (float)bias[bias_id];
        v = fmaxf(v, 0);
        y[id] = (T)v;
    }
}

int AddBiasRelu1(const float* x, const float* bias, float* y, 
                        const int B, const int N, hipStream_t stream) {
    int Num = B * N;
    dim3 block(std::min(Num, 1024));
    dim3 grid((Num + block.x - 1) / block.x);
    AddBiasReluKernel1<float><<<grid, block, 0, stream>>>(x, bias, y, N, Num);
    if (hipGetLastError() != hipSuccess){
        printf("lauch kernel failed");
        return -1;
    }
    return 0;
}

int TestAddBiasRelu(const int B, const int N, const int method_id, const int run_times = 100) {
    GPUTensor x(D_FLOAT32, {B,N}, true);
    GPUTensor bias(D_FLOAT32, {N}, true);
    GPUTensor y(D_FLOAT32, {B,N}, true);
    x.random_uniform_value();
    bias.random_uniform_value();
    y.random_uniform_value();
    hipDeviceSynchronize();

    auto expect_y = (x + bias).relu();
    if (B * N < 256){
        std::cout << "x: " << x;
        std::cout << "bias: " << bias;
        std::cout << "expect y: " << expect_y;
    }

    std::cout << std::endl;
    std::cout << "===== ";
    std::cout << "START method id: " << method_id;
    std::cout <<  "===== " << std::endl;
    if (method_id == 0) {
        AddBiasRelu0(x.data<float>(), bias.data<float>(), y.data<float>(), B, N, nullptr);
        hipDeviceSynchronize();
        std::cout << "y: " << y;
        compare_GPUTensor(expect_y, y);
    } else if (method_id == 1) {
        AddBiasRelu1(x.data<float>(), bias.data<float>(), y.data<float>(), B, N, nullptr);
        hipDeviceSynchronize();
        std::cout << "y: " << y;
        compare_GPUTensor(expect_y, y);        
    }
    std::cout << "===== ";
    std::cout << "END method id: " << method_id;
    std::cout <<  "===== " << std::endl;
    std::cout << std::endl;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipEventBlockingSync);
    std::vector<GPUTensor *> io_list{&x, &bias, &y};
    auto call_fun0 = [&](const std::vector<GPUTensor *> &io_list){
        AddBiasRelu0(io_list[0]->data<float>(), io_list[1]->data<float>(),
                     io_list[2]->data<float>(), B, N, stream);
    };
    auto call_fun1 = [&](const std::vector<GPUTensor *> &io_list){
        AddBiasRelu1(io_list[0]->data<float>(), io_list[1]->data<float>(),
                     io_list[2]->data<float>(), B, N, stream);
    };
    float mem_size_GB = (float)sizeof(float) * (B * N * 2 + N) / (1024 * 1024 * 1024);
    switch (method_id) {
        case 0:
            RunStreamTrueBandwidth(call_fun0, io_list, run_times, stream, mem_size_GB);
            break;
        case 1:
            RunStreamTrueBandwidth(call_fun1, io_list, run_times, stream, mem_size_GB);
            break;
    }
    return 0;
}


int main(int argc, char** argv){
    std::srand(std::time(0));

    int B = 2;
    int N = 5;
    int run_times = 100;
    int device_id = 6;
    int method_id = 0;

    for (int i = 1; i < argc; ++i) {
        std::string opt(argv[i]);
        if (opt == "-B") 
            B = std::stoi(argv[++i]);
        else if (opt == "-N") 
            N = std::stoi(argv[++i]);
        else if (opt == "-R") 
            run_times = std::stoi(argv[++i]);
        else if (opt == "-M") 
            method_id = std::stoi(argv[++i]);
    }
    hipSetDevice(device_id);
    TestAddBiasRelu(B, N, method_id, run_times);


    return 0;
}